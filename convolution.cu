#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "convolution.cuh"

__global__ void kernel_init_noyau(
	/* IN */
	int R,
	float coeff,
	/* OUT */
	int *indi,
	int *indj,
	float *C
	)
{
    // A ECRIRE
}

noyau_t *init_noyau(noyau_t *h_noyau, int R)
{
    // A ECRIRE
}

void free_noyau(noyau_t *h_noyau, noyau_t *d_noyau)
{
    // A ECRIRE
}

__device__ float &elt_ref(void *base_addr, size_t pitch, int i, int j)
{
    float *p_elt = (float*)((char*)base_addr + i*pitch) + j;
    return *p_elt;
}

__global__ void convol_gl(
	noyau_t *d_noyau,
	/* IN */
	float *d_buf_A,
	size_t pitchA,
	int Ni,
	int Nj,
	/* OUT */
	float *d_buf_B,
	size_t pitchB /* IN */
	)
{
    int i = d_noyau->R + blockIdx.x * blockDim.x + threadIdx.x;
    int j = d_noyau->R + blockIdx.y * blockDim.y + threadIdx.y;

    if (i < Ni+d_noyau->R && j < Nj+d_noyau->R)
    {
	float tmp_B = 0;

	for(int k = 0 ; k < d_noyau->KMAX ; k++)
	{
	    const float val_A =
		elt_ref(d_buf_A, pitchA, i+d_noyau->indi[k], j+d_noyau->indj[k]);

	    tmp_B += d_noyau->C[k] * val_A;
	}
	elt_ref(d_buf_B, pitchB, i, j) = tmp_B;
    }
}

__global__ void convol_sh(
	noyau_t *d_noyau,
	/* IN */
	float *d_buf_A,
	size_t pitchA,
	int Ni,
	int Nj,
	/* OUT */
	float *d_buf_B,
	size_t pitchB /* IN */
	)
{
    // A ECRIRE
}

